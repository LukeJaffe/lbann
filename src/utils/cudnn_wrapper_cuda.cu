#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2016, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
//
// cudnn_wrapper .hpp .cpp - cuDNN support - wrapper classes, utility functions
////////////////////////////////////////////////////////////////////////////////

#include "lbann/utils/cudnn_wrapper.hpp"
#include "lbann/utils/exception.hpp"

#include "El.hpp"

using namespace cudnn;
using namespace lbann;

namespace cudnn {

namespace {

__global__ void reduce_kernel(DataType *dst, const DataType *src,
                              El::Int len) {
  int offset = blockIdx.x * blockDim.x + threadIdx.x;
  if (offset >= len) return;
  dst[offset] += src[offset];
}

}

void cudnn_manager::allreduce(const std::vector<DataType*>& gpu_data,
                              El::Int height,
                              El::Int width) {
  if (m_num_gpus < 2) {
    return;
  }

  const El::Int buf_len = 1 << 27;
  const El::Int work_len = buf_len * 2; // double buffering
  const El::Int work_len_bytes = work_len * sizeof(DataType);

  std::vector<DataType*> bufs[2];
  for(int i=0; i<m_num_gpus; ++i) {
    if (get_work_space_size(i) < work_len_bytes) {
      set_work_space_size(i, work_len_bytes); 
    }
    bufs[0].push_back(static_cast<DataType*>(get_work_space(i)));
    bufs[1].push_back(static_cast<DataType*>(get_work_space(i)) + buf_len);
  }  


  El::Int total_len = height * width;
  El::Int offset = 0;

  do {
    El::Int len = std::min(total_len - offset, buf_len);
    int sbuf_idx = 0;
    int dbuf_idx = 1;
    for (int j = 0; j < m_num_gpus - 1; ++j) {
      for(int i = 0; i < m_num_gpus; ++i) {
        CHECK_CUDA(hipSetDevice(m_gpus[i]));
        int src_dev = i;
        int dst_dev = (i + 1) % m_num_gpus;              
        DataType *src_buf = j == 0 ? gpu_data[src_dev] + offset : bufs[sbuf_idx][src_dev];
        DataType *dst_buf = bufs[dbuf_idx][dst_dev];
        // copy to the next device in the ring
        FORCE_CHECK_CUDA(hipMemcpyPeerAsync(dst_buf, dst_dev, src_buf, src_dev,
                                             len * sizeof(DataType), get_stream(src_dev)));
      }
      synchronize();
      for(int i = 0; i < m_num_gpus; ++i) {
        CHECK_CUDA(hipSetDevice(m_gpus[i]));        
        DataType *dst_buf = bufs[dbuf_idx][i];
        // TODO: use Thrust
        int tb_dim = 256;
        int grid_dim = len / tb_dim + (len % tb_dim ? 1 : 0);
        reduce_kernel<<<grid_dim, tb_dim>>>(gpu_data[i] + offset, dst_buf, len);
      }
      std::swap(sbuf_idx, dbuf_idx);
    }
    offset += len;
  } while (offset < total_len);
  
}

#ifdef __LIB_NCCL
/// Convert DataType to NCCL data type. DataType is either double or float (default).
ncclDataType_t nccl_datatype() {
  switch(sizeof(DataTpe) ) {
    case 8:
      return ncclDouble;
    case 4:
      return ncclFloat;
    case 2:
      return ncclHalf;
    default:
      throw lbann::lbann_exception("cudnn_wrapper: invalid data type for NCCL");
  }
}
#endif


void cudnn_manager::allreduce_nccl(const std::vector<DataType*>& gpu_data,
                              El::Int height,
                              El::Int width) {
#ifdef __LIB_NCCL
#define BUFF_LEN	27
/**
  gpu_data is a vector of pointers, each of which points to a part of
  matrix allocated to GPU memory. Since we assume that one MPI rank is
  assigned to one GPU, the number of element in gpu_data is 1. */

  if (m_num_gpus < 2) {
    return;
  }

  /// It is assumed each MPI rank is assigned to one GPU (that is, m_num_gpus==1)
  int local_rank = comm->get_rank_in_node();
  ncclDataType_t type = nccl_datatype();
  El::Int total_len = height * width;

  DataType *target_buffer;
  CHECK_CUDA(hipSetDevice(local_rank));

#if 0

  const El::Int buf_len = 1 << BUFF_LEN;
  El::Int offset = 0;

  FORCE_CHECK_CUDA(hipMalloc((void **) &target_buffer, buf_len*sizeof(DataType)));

  do {
    El::Int len = std::min(total_len - offset, buf_len);

    NCCLCHECK(ncclAllReduce((gpu_data[0]+offset), target_buffer, buf_len, type, ncclSum, m_nccl_comm, get_stream(local_rank)));

    /// Reduction result is stored in target_buffer. Now need to copy the result back to gpu_data.
    FORCE_CHECK_CUDA(hipMemcpy((gpu_data[0]+offset), target_buffer, buf_len*sizeof(DataType),  hipMemcpyDeviceToDevice));
    offset += len;
  } while (offset < total_len);

  FORCE_CHECK_CUDA(hipFree ((void **) &target_buffer));
#else
  FORCE_CHECK_CUDA(hipMalloc((void **) &target_buffer, total_len*sizeof(DataType)));
  NCCLCHECK(ncclAllReduce(gpu_data[0], target_buffer, total_len, type, ncclSum, m_nccl_comm, get_stream(local_rank)));
  FORCE_CHECK_CUDA(hipMemcpy(gpu_data[0], target_buffer, total_len*sizeof(DataType),  hipMemcpyDeviceToDevice));
  FORCE_CHECK_CUDA(hipFree ((void **) &target_buffer));
#endif

#endif
}


} // namespace cudnn
